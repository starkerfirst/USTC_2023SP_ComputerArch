#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define N 1024
#define BLOCK_SIZE 8

// Matrix dimensions
// (chosen as multiples of the thread block size for simplicity)
#define WA N // Matrix A width
#define HA N // Matrix A height
#define WB N // Matrix B width
#define HB WA  // Matrix B height
#define WC WB  // Matrix C width 
#define HC HA  // Matrix C height

//sequential code implemented on cpu
void computeGold(float* C, const float* A, const float* B, unsigned int hA, unsigned int wA, unsigned int wB)
{
    for (unsigned int i = 0; i < hA; ++i)
        for (unsigned int j = 0; j < wB; ++j)
		{
            double sum = 0;
            for (unsigned int k = 0; k < wA; ++k) 
			{
                double a = A[i * wA + k];
                double b = B[k * wB + j];
                sum += a * b;
            }
            C[i * wB + j] = (float)sum;
        }
}

// Initialize a matrix with random float entries.
void randomInit(float* data, int size)
{
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

//Compare the cpu's result with gpu's 
void printDiff(float *data1, float *data2, int width, int height)
{
  int i,j,k;
  int error_count=0;
  for (j=0; j<height; j++) 
  {
    for (i=0; i<width; i++) 
	{
      k = j*width+i;
      if (abs(data1[k] - data2[k]) > 1e-3) 
	  {
         error_count++;
      }
    }
  }
  printf("Total Errors = %d \n", error_count);
}

// matrix multiplication kernel on GPU
__global__ void matrixMul( float* C, float* A, float* B, int wA, int wB, int sign) // sign = 1: using shared memory, sign = 0: not using shared memory
{
     // Declaration of the shared memory array As used to
    // store the sub-matrix of A
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];

    // Declaration of the shared memory array Bs used to
    // store the sub-matrix of B
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

	// Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Index of the first sub-matrix of A processed by the block
    int aBegin = wA * BLOCK_SIZE * by;

    // Index of the last sub-matrix of A processed by the block
    int aEnd   = aBegin + wA - 1;

    // Step size used to iterate through the sub-matrices of A
    int aStep  = BLOCK_SIZE;

    // Index of the first sub-matrix of B processed by the block
    int bBegin = BLOCK_SIZE * bx;

    // Step size used to iterate through the sub-matrices of B
    int bStep  = BLOCK_SIZE * wB;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    float Csub = 0;

    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    if (sign == 1)
    for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) 
	{
        // Load the matrices from device memory
        // to shared memory; each thread loads
        // one element of each matrix
        As[ty][tx] = A[a + wA * ty + tx];
        Bs[ty][tx] = B[b + wB * ty + tx];

        // Synchronize to make sure the matrices are loaded
        __syncthreads();

        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
        for (int k = 0; k < BLOCK_SIZE; ++k)
            Csub += As[ty][k] * Bs[k][tx];

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }
    else
    for (int a = aBegin, b = bBegin; a <= aEnd; a += aStep, b += bStep) 
    {
        for (int k = 0; k < BLOCK_SIZE; ++k)
            Csub += A[a + wA * ty + k] * B[b + wB * k + tx];

        // Synchronize to make sure that the preceding
        // computation is done before loading two new
        // sub-matrices of A and B in the next iteration
        __syncthreads();
    }

    // Write the block sub-matrix to device memory;
    // each thread writes one element
    int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    C[c + wB * ty + tx] = Csub;
}


int main(int argc, char **argv)
{
	// set seed for rand()
    srand((unsigned)time(NULL));

    // allocate host memory for matrices A and B
    unsigned int size_A = WA * HA;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float* h_A = (float*) malloc(mem_size_A);
    unsigned int size_B = WB * HB;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float* h_B = (float*) malloc(mem_size_B);
    unsigned int size_C = WC * HC;
    unsigned int mem_size_C = sizeof(float) * size_C;
    float* d_C;

    // initialize host memory
    randomInit(h_A, size_A);
    randomInit(h_B, size_B);

    // CPU
    // compute reference solution
    float* reference = (float*) malloc(mem_size_C);
    double cpu_timer = 0;
    double begin,end;
    begin = clock();
    computeGold(reference, h_A, h_B, HA, WA, WB);
    end = clock();
    cpu_timer = (double)(end - begin)/CLOCKS_PER_SEC;

    // print timers
    printf("CPU Processing time: %f (s) \n", cpu_timer);

    // GPU
    // allocate device memory
    float* d_A;
    hipMalloc((void**) &d_A, mem_size_A);
    float* d_B;
    hipMalloc((void**) &d_B, mem_size_B);

    // copy host memory to device
    hipMemcpy(d_A, h_A, mem_size_A, hipMemcpyHostToDevice) ;
    hipMemcpy(d_B, h_B, mem_size_B, hipMemcpyHostToDevice) ;

    // allocate device memory for result
    hipMalloc((void**) &d_C, mem_size_C);

    // allocate host memory for the result
    float* h_C = (float*) malloc(mem_size_C);
    
    // create and start gpu_timer by cuda_event
    float gpu_timer = 0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // setup execution parameters
    dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
    dim3 grid(WC / threads.x, HC / threads.y);

    // execute the kernel -- mode 1
    hipEventRecord(start, 0);
    matrixMul<<< grid, threads >>>(d_C, d_A, d_B, WA, WB, 1);
    hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

    // stop and destroy gpu_timer
    hipEventElapsedTime(&gpu_timer, start, stop);
    printf("GPU Processing time with shared mem: %f (s) \n", gpu_timer/1000);

    // copy result from device to host
    hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);
    printDiff(reference, h_C, WC, HC);

    // execute the kernel -- mode 1
    hipEventRecord(start, 0);
    matrixMul<<< grid, threads >>>(d_C, d_A, d_B, WA, WB, 0);
    hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

    // stop and destroy gpu_timer
    hipEventElapsedTime(&gpu_timer, start, stop);
    printf("GPU Processing time without shared mem: %f (s) \n", gpu_timer/1000);

    // copy result from device to host
    hipMemcpy(h_C, d_C, mem_size_C, hipMemcpyDeviceToHost);
    printDiff(reference, h_C, WC, HC);

    // clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
    free(reference);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipEventDestroy(start);
    hipEventDestroy(stop);

}